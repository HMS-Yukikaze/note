#include "hip/hip_runtime.h"
#include "hello.cuh"


__global__ void hello_from_gpu(){

    const int bx = blockIdx.x;
    const int by = blockIdx.y;
    const int bz = blockIdx.z;

    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int tz = threadIdx.z;

    printf("gpu: hello world! block(%d, %d, %d) -- thread(%d, %d, %d)\n", bx, by, bz, tx, ty, tz);
}

void launch_hello_from_gpu() {
    dim3 threadsPerBlock(2, 2, 2);
    dim3 numBlocks(2, 2, 2);

    hello_from_gpu<<<numBlocks, threadsPerBlock>>>();
    hipDeviceSynchronize();
}
